#include "hip/hip_runtime.h"
#include "MgGeneratePoisson.h"
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#define BLOCKSIZE 1024

__global__ void ConvertUnsigedToDouble(double* arrayDouble, unsigned* arrayUnsigned, size_t N)
{
	size_t id = threadIdx.x + blockDim.x * blockIdx.x;

	if (id < N)
	{
		arrayDouble[id] = (double)arrayUnsigned[id];
	}
}

// Same lambda, use host API
void MgGeneratePoissonSameLambda(double* resultArray, size_t resultSize, double lambda, unsigned long long seed)
{
	// generate Poisson random number of unsigned type
	unsigned* resultArrayUint_dev;
	double* resultArrayDouble_dev;

	hipMalloc(&resultArrayUint_dev, sizeof(unsigned) * resultSize);
	hipMalloc(&resultArrayDouble_dev, sizeof(double) * resultSize);

	hiprandGenerator_t gen;

	hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);

	hiprandSetPseudoRandomGeneratorSeed(gen, seed);

	hiprandGeneratePoisson(gen, resultArrayUint_dev, resultSize, lambda);

	hiprandDestroyGenerator(gen);

	hipDeviceSynchronize();

	// convert unsigend type into double type
	ConvertUnsigedToDouble << <(resultSize + BLOCKSIZE - 1) / BLOCKSIZE, BLOCKSIZE >> > (resultArrayDouble_dev, resultArrayUint_dev, resultSize);
	hipDeviceSynchronize();

	// copy result back to hose
	hipMemcpy(resultArray, resultArrayDouble_dev, resultSize * sizeof(double), hipMemcpyDeviceToHost);

	hipFree(resultArrayUint_dev);
	hipFree(resultArrayDouble_dev);
}


__global__ void InitializeState(hiprandState* state, size_t N, unsigned long long seed)
{
	size_t id = threadIdx.x + blockDim.x * blockIdx.x;

	if (id < N)
	{
		hiprand_init(seed, id, 0, &state[id]);
	}
}

__global__ void GiveRandomNumbersToVariousLambda(double* resultArrayDev, double* lambdaDev, size_t N, hiprandState* state)
{
	hiprandState localState = state[threadIdx.x];

	int loops = (N + BLOCKSIZE - 1) / BLOCKSIZE;
	int id;
	for (int k = 0; k < loops; k++)
	{
		id = threadIdx.x + BLOCKSIZE * k;
		if(id < N)
			resultArrayDev[id] = (double)hiprand_poisson(&localState, lambdaDev[id]);
	}
}

// Various lambda, use device API
void MgGeneratePoissonVariousLambda(double* resultArray, double* lambda, size_t resultSize, unsigned long long seed)
{
	double* resultArray_dev;
	hipMalloc((void**)& resultArray_dev, resultSize * sizeof(double));

	// initialize state
	hiprandState* devState;
	hipMalloc((void**)& devState, BLOCKSIZE * sizeof(hiprandState));
	InitializeState <<<1, BLOCKSIZE>>> (devState, BLOCKSIZE, seed);
	hipDeviceSynchronize();

	// copy data of lamda to device
	double* lambda_dev;
	hipMalloc((void**)& lambda_dev, resultSize * sizeof(double));
	hipMemcpy(lambda_dev, lambda, resultSize * sizeof(double), hipMemcpyHostToDevice);
	hipDeviceSynchronize();

	// get random numbers
	GiveRandomNumbersToVariousLambda <<<1, BLOCKSIZE >>> (resultArray_dev, lambda_dev, resultSize, devState);
	hipDeviceSynchronize();

	hipMemcpy(resultArray, resultArray_dev, resultSize * sizeof(double), hipMemcpyDeviceToHost);
	hipDeviceSynchronize();

	hipFree(lambda_dev);
	hipFree(devState);
	hipFree(resultArray_dev);
}